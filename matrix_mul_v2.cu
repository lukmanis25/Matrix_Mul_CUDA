#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h> 

typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

#define FIXED_VALUE 2.0f
#define BLOCK_SIZE 16

__device__ float GetElement(const Matrix mat, int row, int col)
{
    return mat.elements[row * mat.stride + col];
}

__device__ void SetElement(Matrix mat, int row, int col, float value)
{
    mat.elements[row * mat.stride + col] = value;
}

 __device__ Matrix GetSubMatrix(const Matrix mat, int subRow, int subCol)
{
    Matrix subMat;
    subMat.width = BLOCK_SIZE;
    subMat.height = BLOCK_SIZE;
    subMat.stride = mat.stride;
    subMat.elements = &mat.elements[subRow * BLOCK_SIZE * mat.stride + subCol * BLOCK_SIZE];
    return subMat;
}


 __global__ void CalcMatMulKernel(Matrix matA, Matrix matB, Matrix matC)
{
    int blockRowIdx = blockIdx.y;
    int blockColIdx = blockIdx.x;

    Matrix subMatC = GetSubMatrix(matC, blockRowIdx, blockColIdx);

    float result = 0.0f;

    int localRowIdx = threadIdx.y;
    int localColIdx = threadIdx.x;

    for (int phase = 0; phase < (matA.width / BLOCK_SIZE); ++phase) {
        Matrix subMatA = GetSubMatrix(matA, blockRowIdx, phase);
        Matrix subMatB = GetSubMatrix(matB, phase, blockColIdx);

        __shared__ float sharedMatA[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float sharedMatB[BLOCK_SIZE][BLOCK_SIZE];

        sharedMatA[localRowIdx][localColIdx] = GetElement(subMatA, localRowIdx, localColIdx);
        sharedMatB[localRowIdx][localColIdx] = GetElement(subMatB, localRowIdx, localColIdx);

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            result += sharedMatA[localRowIdx][k] * sharedMatB[k][localColIdx];
        }
        __syncthreads();
    }

    SetElement(subMatC, localRowIdx, localColIdx, result);
}

void MatMul(const Matrix matA, const Matrix matB, Matrix matC)
{
    int paddedWidthA = ((matA.width + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    int paddedHeightA = ((matA.height + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    int paddedWidthB = ((matB.width + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    int paddedHeightB = ((matB.height + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    int paddedWidthC = ((matC.width + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    int paddedHeightC = ((matC.height + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;

    Matrix devMatA;
    devMatA.width = paddedWidthA;
    devMatA.height = paddedHeightA;
    devMatA.stride = paddedWidthA;
    size_t bytesA = paddedWidthA * paddedHeightA * sizeof(float);
    hipMalloc(&devMatA.elements, bytesA);
    hipMemset(devMatA.elements, 0, bytesA);
    
    Matrix devMatB;
    devMatB.width = paddedWidthB;
    devMatB.height = paddedHeightB;
    devMatB.stride = paddedWidthB;
    size_t bytesB = paddedWidthB * paddedHeightB * sizeof(float);
    hipMalloc(&devMatB.elements, bytesB);
    hipMemset(devMatB.elements, 0, bytesB);

    Matrix devMatC;
    devMatC.width = paddedWidthC;
    devMatC.height = paddedHeightC;
    devMatC.stride = paddedWidthC;
    size_t bytesC = paddedWidthC * paddedHeightC * sizeof(float);
    hipMalloc(&devMatC.elements, bytesC);
    hipMemset(devMatC.elements, 0, bytesC);

    for (int i = 0; i < matA.height; ++i) {
        hipMemcpy(&devMatA.elements[i * devMatA.stride], &matA.elements[i * matA.width], matA.width * sizeof(float), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < matB.height; ++i) {
        hipMemcpy(&devMatB.elements[i * devMatB.stride], &matB.elements[i * matB.width], matB.width * sizeof(float), hipMemcpyHostToDevice);
    }

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((paddedWidthC + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (paddedHeightC + BLOCK_SIZE - 1) / BLOCK_SIZE);

    CalcMatMulKernel<<<blocksPerGrid, threadsPerBlock>>>(devMatA, devMatB, devMatC);

    for (int i = 0; i < matC.height; ++i) {
        hipMemcpy(&matC.elements[i * matC.width], &devMatC.elements[i * devMatC.stride], matC.width * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(devMatA.elements);
    hipFree(devMatB.elements);
    hipFree(devMatC.elements);
}




/***************
**TESTING CODE**
****************/
void fillMatrixRandom(Matrix &matrix) {
    for (int i = 0; i < matrix.width * matrix.height; i++) {
        matrix.elements[i] = static_cast<float>(rand()) / RAND_MAX; // Losowe wartości między 0 a 1
    }
}

void fillMatrixFixed(Matrix &matrix, float value) {
    for (int i = 0; i < matrix.width * matrix.height; i++) {
        matrix.elements[i] = value;
    }
}

void printMatrix(const Matrix &matrix) {
    for (int i = 0; i < matrix.height; i++) {
        for (int j = 0; j < matrix.width; j++) {
            printf("%f ", matrix.elements[i * matrix.width + j]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[]) {
    if (argc < 4 || argc > 6) {
        printf("Użycie: %s <n> <m> <k> [print] [fixed]\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    int k = atoi(argv[3]);
    bool printResult = (argc >= 5 && strcmp(argv[4], "print") == 0);
    bool useFixedValues = (argc == 6 && strcmp(argv[5], "fixed") == 0);

    Matrix A, B, C;
    A.width = m; A.height = n;
    B.width = k; B.height = m;
    C.width = k; C.height = n;

    A.elements = (float*)malloc(A.width * A.height * sizeof(float));
    B.elements = (float*)malloc(B.width * B.height * sizeof(float));
    C.elements = (float*)malloc(C.width * C.height * sizeof(float));

    srand(time(0));
    if (useFixedValues) {
        fillMatrixFixed(A, FIXED_VALUE);
        fillMatrixFixed(B, FIXED_VALUE);
    } else {
        fillMatrixRandom(A);
        fillMatrixRandom(B);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    MatMul(A, B, C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Czas wykonania mnożenia macierzy na GPU: %f ms\n", milliseconds);
    printf("Wymiary wynikowa macierz C: %d x %d\n", C.height, C.width);
    if (printResult) {
        printMatrix(C);
    }

    free(A.elements);
    free(B.elements);
    free(C.elements);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}