#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h> 

typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

#define FIXED_VALUE 2.0f
#define BLOCK_SIZE 16

__device__ float GetElement(const Matrix mat, int row, int col)
{
    return mat.elements[row * mat.stride + col];
}

__device__ void SetElement(Matrix mat, int row, int col, float value)
{
    mat.elements[row * mat.stride + col] = value;
}

 __device__ Matrix GetSubMatrix(const Matrix mat, int subRow, int subCol)
{
    Matrix subMat;
    subMat.width = BLOCK_SIZE;
    subMat.height = BLOCK_SIZE;
    subMat.stride = mat.stride;
    subMat.elements = &mat.elements[subRow * BLOCK_SIZE * mat.stride + subCol * BLOCK_SIZE];
    return subMat;
}


 __global__ void CalcMatMulKernel(Matrix matA, Matrix matB, Matrix matC)
{
    int blockRowIdx = blockIdx.y;
    int blockColIdx = blockIdx.x;

    Matrix subMatC = GetSubMatrix(matC, blockRowIdx, blockColIdx);

    float result = 0.0f;

    int localRowIdx = threadIdx.y;
    int localColIdx = threadIdx.x;

    for (int phase = 0; phase < (matA.width / BLOCK_SIZE); ++phase) {
        Matrix subMatA = GetSubMatrix(matA, blockRowIdx, phase);
        Matrix subMatB = GetSubMatrix(matB, phase, blockColIdx);

        __shared__ float sharedMatA[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float sharedMatB[BLOCK_SIZE][BLOCK_SIZE];

        sharedMatA[localRowIdx][localColIdx] = GetElement(subMatA, localRowIdx, localColIdx);
        sharedMatB[localRowIdx][localColIdx] = GetElement(subMatB, localRowIdx, localColIdx);

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            result += sharedMatA[localRowIdx][k] * sharedMatB[k][localColIdx];
        }
        __syncthreads();
    }

    SetElement(subMatC, localRowIdx, localColIdx, result);
}

void MatMul(const Matrix matA, const Matrix matB, Matrix matC)
{
    Matrix devMatA;
    devMatA.width = devMatA.stride = matA.width;
    devMatA.height = matA.height;
    size_t bytesA = matA.width * matA.height * sizeof(float);
    hipMalloc(&devMatA.elements, bytesA);
    hipMemcpy(devMatA.elements, matA.elements, bytesA, hipMemcpyHostToDevice);

    Matrix devMatB;
    devMatB.width = devMatB.stride = matB.width;
    devMatB.height = matB.height;
    size_t bytesB = matB.width * matB.height * sizeof(float);
    hipMalloc(&devMatB.elements, bytesB);
    hipMemcpy(devMatB.elements, matB.elements, bytesB, hipMemcpyHostToDevice);

    Matrix devMatC;
    devMatC.width = devMatC.stride = matC.width;
    devMatC.height = matC.height;
    size_t bytesC = matC.width * matC.height * sizeof(float);
    hipMalloc(&devMatC.elements, bytesC);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((matB.width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (matA.height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    CalcMatMulKernel<<<blocksPerGrid, threadsPerBlock>>>(devMatA, devMatB, devMatC);

    hipMemcpy(matC.elements, devMatC.elements, bytesC, hipMemcpyDeviceToHost);

    hipFree(devMatA.elements);
    hipFree(devMatB.elements);
    hipFree(devMatC.elements);
}




/***************
**TESTING CODE**
****************/
void fillMatrixRandom(Matrix &matrix) {
    for (int i = 0; i < matrix.width * matrix.height; i++) {
        matrix.elements[i] = static_cast<float>(rand()) / RAND_MAX; // Losowe wartości między 0 a 1
    }
}

void fillMatrixFixed(Matrix &matrix, float value) {
    for (int i = 0; i < matrix.width * matrix.height; i++) {
        matrix.elements[i] = value;
    }
}

void printMatrix(const Matrix &matrix) {
    for (int i = 0; i < matrix.height; i++) {
        for (int j = 0; j < matrix.width; j++) {
            printf("%f ", matrix.elements[i * matrix.width + j]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[]) {
    if (argc < 4 || argc > 6) {
        printf("Użycie: %s <n> <m> <k> [print] [fixed]\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    int k = atoi(argv[3]);
    bool printResult = (argc >= 5 && strcmp(argv[4], "print") == 0);
    bool useFixedValues = (argc == 6 && strcmp(argv[5], "fixed") == 0);

    Matrix A, B, C;
    A.width = m; A.height = n;
    B.width = k; B.height = m;
    C.width = k; C.height = n;

    A.elements = (float*)malloc(A.width * A.height * sizeof(float));
    B.elements = (float*)malloc(B.width * B.height * sizeof(float));
    C.elements = (float*)malloc(C.width * C.height * sizeof(float));

    srand(time(0));
    if (useFixedValues) {
        fillMatrixFixed(A, FIXED_VALUE);
        fillMatrixFixed(B, FIXED_VALUE);
    } else {
        fillMatrixRandom(A);
        fillMatrixRandom(B);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    MatMul(A, B, C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Czas wykonania mnożenia macierzy na GPU: %f ms\n", milliseconds);
    printf("Wymiary wynikowa macierz C: %d x %d\n", C.height, C.width);
    if (printResult) {
        printMatrix(C);
    }

    free(A.elements);
    free(B.elements);
    free(C.elements);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}