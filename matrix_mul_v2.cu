#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h> 


// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

#define FIXED_VALUE 2.0f
#define BLOCK_SIZE 16

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}
// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}
// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}
// Thread block size
#define BLOCK_SIZE 16
// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);
    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);
    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;
    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;
    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

/***************
**TESTING CODE**
****************/
void fillMatrixRandom(Matrix &matrix) {
    for (int i = 0; i < matrix.width * matrix.height; i++) {
        matrix.elements[i] = static_cast<float>(rand()) / RAND_MAX; // Losowe wartości między 0 a 1
    }
}

void fillMatrixFixed(Matrix &matrix, float value) {
    for (int i = 0; i < matrix.width * matrix.height; i++) {
        matrix.elements[i] = value;
    }
}

void printMatrix(const Matrix &matrix) {
    for (int i = 0; i < matrix.height; i++) {
        for (int j = 0; j < matrix.width; j++) {
            printf("%f ", matrix.elements[i * matrix.width + j]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[]) {
    if (argc < 4 || argc > 6) {
        printf("Użycie: %s <n> <m> <k> [print] [fixed]\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    int k = atoi(argv[3]);
    bool printResult = (argc >= 5 && strcmp(argv[4], "print") == 0);
    bool useFixedValues = (argc == 6 && strcmp(argv[5], "fixed") == 0);

    Matrix A, B, C;
    A.width = m; A.height = n;
    B.width = k; B.height = m;
    C.width = k; C.height = n;

    A.elements = (float*)malloc(A.width * A.height * sizeof(float));
    B.elements = (float*)malloc(B.width * B.height * sizeof(float));
    C.elements = (float*)malloc(C.width * C.height * sizeof(float));

    srand(time(0));
    if (useFixedValues) {
        fillMatrixFixed(A, FIXED_VALUE);
        fillMatrixFixed(B, FIXED_VALUE);
    } else {
        fillMatrixRandom(A);
        fillMatrixRandom(B);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    MatMul(A, B, C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Czas wykonania mnożenia macierzy na GPU: %f ms\n", milliseconds);
    printf("Wymiary wynikowa macierz C: %d x %d\n", C.height, C.width);
    if (printResult) {
        printMatrix(C);
    }

    free(A.elements);
    free(B.elements);
    free(C.elements);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}